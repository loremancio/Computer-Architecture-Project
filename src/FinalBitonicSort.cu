
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <cmath>
using namespace std;

/* 
  arrayInit fill the array with random values 
  it takes as input the array to fill and its lenght  
*/
void arrayInit(int *arr, unsigned long length) {
    unsigned long i;
    for (i = 0; i < length; ++i) {
        arr[i] = (rand() % 100);
    }
}
/* 
  checkArray check if the array after the bitonic sort is properly sorted 
  it takes as input the array to fill and its lenght
  return true if the array is sorted, false otherwise
  */
bool checkArray(int *arr, unsigned long long length) {
    for (unsigned long long i = 1; i < length; ++i) {
        if(arr[i-1]>arr[i]){
            printf("Array not sorted");
            return false;
        }
    }
    return true;
}

/* 
  swap_values use a temp variable in order to swap two values
  it is performed in GPU and it is used by bitonic_sort_kernel
  it takes as input the two values to swap 
  */
__device__ void swap_values(int *a, int *b){
  int temp = *a;
  *a = *b;
  *b = temp;
}

/* 
 bitonic_sort_kernel performs a single pass of sorting in the GPU 
 it takes as input the array on which to operate, the distance between the element to
 swap in case the check is passed and the size of the sequence to sort
*/
__global__ void bitonic_sort_kernel(int *arr, unsigned long long distance, unsigned long long subSequence_size){
  //get the thread id and the check value using the xor operands
  unsigned long long i, xorCheck; 
  i = threadIdx.x + blockDim.x * blockIdx.x;
  
  //get the element in the array to sort
  xorCheck = i^distance;

  //Sort only the element that are distant enough
  if ((xorCheck)>i) {
    
    //if the operator produce 0 we are in the ascending part
    //of the bitonic sequence
    if ((i & subSequence_size)==0) {
      if (arr[i]>arr[xorCheck]) {
        swap_values(&arr[i],&arr[xorCheck]);
      }
    }

    //otherwise we are in the decending part
    else {
      if (arr[i]<arr[xorCheck]) {
        swap_values(&arr[i],&arr[xorCheck]);
      }
    }
  }
}

/* 
  bitonic_sort performs the operations to sort the array 
  it takes as input the array to sort and its lenght
  */
void bitonic_sort(int *arr, unsigned long long array_size, int numThreads){
    int *cuda_arr; // device array
    size_t size = array_size * sizeof(int); // size * 4 byte
    
    //allocate memory on device
    hipMalloc((void**) &cuda_arr, size); 
    //copy the original array to the device one
    hipMemcpy(cuda_arr, arr, size, hipMemcpyHostToDevice); 

    //set the number of threads per blocks and calculate the number of blocks
    int block_dim=(array_size + numThreads - 1) / numThreads;

    
    unsigned long long distance, subSequence_size; 

    //initialize the time recording without cudamemcpy
    hipEvent_t start, stop;
    float elapsed;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //iterate through the array sorting sequence of ascending dimention over distances
    //of decending dimention
    for (subSequence_size = 2; subSequence_size <= array_size; subSequence_size <<= 1) 
      for (distance = subSequence_size >> 1; distance > 0; distance = distance >> 1)        
        bitonic_sort_kernel<<<block_dim, numThreads>>>(cuda_arr, distance, subSequence_size);
        
    //calculate elapsed time withouth the cudaMemcpy
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("%f,", elapsed);
    
    //copy back to the host array
    hipMemcpy(arr, cuda_arr, size, hipMemcpyDeviceToHost); 
    //free device memory
    hipFree(cuda_arr); 
   
}

int main(int argc, char** argv){


  int arraySize = pow(2, atoi(argv[1]));  // Size of the array (2^21 elements
  unsigned int numThreads = atoi(argv[2]);  // Number of threads
  //if the number of threads is not a power of 2 return an error
  if ((numThreads & (numThreads - 1)) != 0) {
    printf("The number of threads must be a power of 2\n");
    return 1;
  }

  int *arr = new int[arraySize];

  srand(time(0));

  for (int i = 0; i < arraySize; i++) {
      arr[i] = rand() % 2097152;
  }



  hipEvent_t startExt, outExt;
  //contains the elapsed time on the bitonic sort with bitonic sort
  float elapsed; 
  hipEventCreate(&startExt);
  hipEventCreate(&outExt);
  hipEventRecord(startExt, 0);
  //=========================================//
  bitonic_sort(arr, arraySize, numThreads);
  //=========================================//
  hipEventRecord(outExt, 0);
  hipEventSynchronize(outExt);
  hipEventElapsedTime(&elapsed, startExt, outExt);
  hipEventDestroy(startExt);
  hipEventDestroy(outExt);


  printf("%f\n", elapsed);
}