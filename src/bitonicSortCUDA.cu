
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <cmath>
using namespace std;

/* 
in-device function to swap two elements in the array
*/
__device__ void swap(int *a, int *b){
  int tmp = *a;
  *a = *b;
  *b = tmp;
}

/* 
bitonicKernel utilizes the GPU to perform a single sorting pass on an array. 
It accepts three inputs: the array to be sorted, 
the distance between elements that may be swapped based on a specific check, 
and the size of the sequence to be sorted.
*/
__global__ void bitonicKernel(int *arr, unsigned long long distance, unsigned long long subSequence_size) {
  unsigned long long i, xorCheck;

  // Thread identification and index calculation
  i = threadIdx.x + blockDim.x * blockIdx.x;

  // Calculate bitonic comparison element using XOR
  xorCheck = i ^ distance;

  // Check if current element needs to be compared (based on bitonic sequence)
  if (xorCheck > i) {
    // Check if current element is at the beginning of a subsequence
    if ((i & subSequence_size) == 0) {
      // Perform ascending comparison for the first half of the bitonic sequence
      if (arr[i] > arr[xorCheck]) {
        swap(&arr[i], &arr[xorCheck]);
      }
    } else {
      // Perform descending comparison for the second half of the bitonic sequence
      if (arr[i] < arr[xorCheck]) {
        swap(&arr[i], &arr[xorCheck]);
      }
    }
  }
}


/* 
bitonicSort sets up the GPU and calls the bitonicKernel function to sort an array.
*/
void bitonicSort(int *arr, unsigned long long array_size, int num_threads) {
  // Allocate memory on the GPU for the array
  int *cuda_arr;
  size_t size = array_size * sizeof(int);
  hipMalloc((void**) &cuda_arr, size);

  // Copy the array from host to device memory
  hipMemcpy(cuda_arr, arr, size, hipMemcpyHostToDevice);

  // Calculate the block size for efficient thread utilization
  int block_dim = (array_size + num_threads - 1) / num_threads;

  // Variables for bitonic sort parameters (can be pre-calculated for efficiency)
  unsigned long long distance, subSequence_size;

  // Timing variables for performance measurement
  hipEvent_t start, stop;
  float elapsed;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);  // Record start time

  // Nested loop for bitonic sort passes
  for (subSequence_size = 2; subSequence_size <= array_size; subSequence_size <<= 1) {
    for (distance = subSequence_size >> 1; distance > 0; distance >>= 1) {
      // Launch the bitonicKernel with appropriate block and thread configuration
      bitonicKernel<<<block_dim, num_threads>>>(cuda_arr, distance, subSequence_size);
    }
  }

  // Record end time and synchronize for accurate measurement
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Print elapsed time in milliseconds (assuming milliseconds desired)
  printf("%f,", elapsed / 1000);  // Adjust division factor based on desired unit

  // Copy the sorted array back from device to host memory
  hipMemcpy(arr, cuda_arr, size, hipMemcpyDeviceToHost);

  // Free the allocated GPU memory
  hipFree(cuda_arr);
}

int main(int argc, char** argv) {
  // Check for valid program arguments
  if (argc != 3) {
    printf("Usage: %s <array_size> <num_threads>\n", argv[0]);
    return 1;
  }

  // Get array size from command-line argument
  int array_size = pow(2, atoi(argv[1]));
  if (array_size <= 0) {
    printf("Array size must be a positive power of 2\n");
    return 1;
  }

  // Get number of threads from command-line argument
  unsigned int num_threads = atoi(argv[2]);

  // Validate that num_threads is a power of 2
  if ((num_threads & (num_threads - 1)) != 0) {
    printf("The number of threads must be a power of 2\n");
    return 1;
  }

  // Allocate memory on host (CPU) for the array
  int *arr = new int[array_size];

  // Seed random number generator
  srand(time(0));

  // Initialize array with random values (0 to 2097151)
  for (int i = 0; i < array_size; i++) {
    arr[i] = rand() % 2097152;
  }

  // Timing setup for overall execution time (including memory transfers)
  hipEvent_t start_exec, end_exec;
  float elapsed;
  hipEventCreate(&start_exec);
  hipEventCreate(&end_exec);
  hipEventRecord(start_exec, 0);  // Record start time

  // Call bitonicSort function for sorting
  bitonicSort(arr, array_size, num_threads);

  // Record end time and synchronize for accurate measurement
  hipEventRecord(end_exec, 0);
  hipEventSynchronize(end_exec);
  hipEventElapsedTime(&elapsed, start_exec, end_exec);
  hipEventDestroy(start_exec);
  hipEventDestroy(end_exec);

  // Print total execution time in milliseconds
  printf("%f\n", elapsed / 1000);

  // Deallocate host memory
  delete[] arr;

  return 0;
}